#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include "atomic.h"
#include "cuda_common.h"
#include "tensor_ops.h"

namespace gs {
namespace impl {

template <typename IdType>
__global__ void _ListSamplingIndexKernel(const uint64_t rand_seed,
                                         const IdType *const in_data,
                                         int64_t *const out_index,
                                         const int num_items,
                                         const int num_picks) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tot = blockDim.x * gridDim.x;

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  for (int64_t idx = num_picks + tid; idx < num_items; idx += tot) {
    int64_t num = hiprand(&rng) % (idx + 1);
    if (num < num_picks) {
      AtomicMax(out_index + num, idx);
    }
  }
}

/**
 * @brief ListSampling, using A-Res sampling for replace = False and uniform
 * sampling for replace = True. It will return (selected_data, selected_index)
 *
 * @tparam IdType
 * @param data
 * @param num_picks
 * @param replace
 * @return std::tuple<torch::Tensor, torch::Tensor>
 */
template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor> _ListSampling(torch::Tensor data,
                                                       int64_t num_picks,
                                                       bool replace) {
  int num_items = data.numel();
  torch::TensorOptions index_options =
      torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);

  if (num_items <= num_picks and !replace) {
    // todo (ping), do we need clone here?
    return std::make_tuple(data.clone(),
                           torch::arange(num_items, index_options));
  }

  torch::Tensor select;
  torch::Tensor index;

  if (replace) {
    select = torch::empty(num_picks, data.options());
    index = torch::empty(num_picks, index_options);

    uint64_t random_seed = 7777;
    using it = thrust::counting_iterator<IdType>;
    thrust::for_each(
        it(0), it(num_picks),
        [in = data.data_ptr<IdType>(), out_data = select.data_ptr<IdType>(),
         out_index = index.data_ptr<int64_t>(), num_items, num_picks,
         random_seed] __device__(IdType i) mutable {
          hiprandState rng;
          hiprand_init(i * random_seed, 0, 0, &rng);
          int64_t _id = hiprand(&rng) % num_items;
          out_data[i] = in[_id];
          out_index[i] = _id;
        });

  } else {
    index = torch::arange(num_picks, index_options);

    uint64_t random_seed = 7777;
    constexpr int BLOCK_SIZE = 256;
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((num_items + BLOCK_SIZE - 1) / BLOCK_SIZE);
    _ListSamplingIndexKernel<IdType>
        <<<grid, block>>>(random_seed, data.data_ptr<IdType>(),
                          index.data_ptr<int64_t>(), num_items, num_picks);

    select = data.index({index});
  }

  return std::make_tuple(select, index);
}

std::tuple<torch::Tensor, torch::Tensor> ListSampling(torch::Tensor data,
                                                      int64_t num_picks,
                                                      bool replace) {
  return _ListSampling<int64_t>(data, num_picks, replace);
}

}  // namespace impl

}  // namespace gs
