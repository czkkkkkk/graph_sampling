#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include "atomic.h"
#include "cuda_common.h"
#include "tensor_ops.h"

namespace gs {
namespace impl {

/**
 * @brief ListSampling, using A-Res sampling for replace = False and uniform
 * sampling for replace = True. It will return (selected_data, selected_index)
 *
 * @tparam IdType
 * @param data
 * @param num_picks
 * @param replace
 * @return std::tuple<torch::Tensor, torch::Tensor>
 */
template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor> _ListSampling(torch::Tensor data,
                                                       int64_t num_picks,
                                                       bool replace) {
  int num_items = data.numel();
  torch::TensorOptions index_options =
      torch::TensorOptions().dtype(torch::kInt64).device(torch::kCUDA);

  if (num_items <= num_picks and !replace) {
    // todo (ping), do we need clone here?
    return std::make_tuple(data.clone(),
                           torch::arange(num_items, index_options));
  }

  torch::Tensor select;
  torch::Tensor index;

  if (replace) {
    select = torch::empty(num_picks, data.options());
    index = torch::empty(num_picks, index_options);

    uint64_t random_seed = 7777;
    using it = thrust::counting_iterator<IdType>;
    thrust::for_each(
        it(0), it(num_picks),
        [in = data.data_ptr<IdType>(), out_data = select.data_ptr<IdType>(),
         out_index = index.data_ptr<int64_t>(), num_items, num_picks,
         random_seed] __device__(IdType i) mutable {
          hiprandState rng;
          hiprand_init(i * random_seed, 0, 0, &rng);
          int64_t _id = hiprand(&rng) % num_items;
          out_data[i] = in[_id];
          out_index[i] = _id;
        });

  } else {
    index = torch::arange(num_picks, index_options);

    uint64_t random_seed = 7777;
    using it = thrust::counting_iterator<IdType>;
    thrust::for_each(it(num_picks), it(num_items),
                     [out_index = index.data_ptr<int64_t>(), num_picks,
                      random_seed] __device__(IdType idx) mutable {
                       if (idx < num_picks) {
                         return;
                       }
                       hiprandState rng;
                       hiprand_init(idx * random_seed, 0, 0, &rng);
                       int64_t num = hiprand(&rng) % (idx + 1);
                       if (num < num_picks) {
                         AtomicMax(out_index + num, idx);
                       }
                     });

    select = data.index({index});
  }

  return std::make_tuple(select, index);
}

std::tuple<torch::Tensor, torch::Tensor> ListSamplingCUDA(torch::Tensor data,
                                                          int64_t num_picks,
                                                          bool replace) {
  return _ListSampling<int64_t>(data, num_picks, replace);
}

}  // namespace impl

}  // namespace gs
