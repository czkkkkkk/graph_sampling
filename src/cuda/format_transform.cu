#include "hip/hip_runtime.h"
#include "graph_ops.h"

#include "cuda_common.h"
#include "utils.h"

namespace gs {
namespace impl {

/*!
 * @brief Repeat elements.
 *
 * @param pos: The position of the output buffer to write the value.
 * @param out: Output buffer.
 * @param n_col: Length of positions
 * @param length: Number of values
 *
 * For example:
 * pos = [0, 1, 3, 4]
 * (implicit) val = [0, 1, 2]
 * then,
 * out = [0, 1, 1, 2]
 */
template <typename IdType, bool UseNMap>
__global__ void _RepeatKernel(const IdType* pos, const IdType* NIDMap,
                              IdType* out, int64_t n_col, int64_t length) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    IdType i = hipcub::UpperBound(pos, n_col, tx) - 1;
    out[tx] = UseNMap ? NIDMap[i] : i;
    tx += stride_x;
  }
}

std::pair<torch::Tensor, torch::Tensor> CSC2COOCUDA(torch::Tensor indptr,
                                                    torch::Tensor indices) {
  auto coo_size = indices.numel();
  auto col = torch::zeros(coo_size, indptr.options());

  dim3 block(128);
  dim3 grid((coo_size + block.x - 1) / block.x);
  _RepeatKernel<int64_t, false>
      <<<grid, block>>>(indptr.data_ptr<int64_t>(), nullptr,
                        col.data_ptr<int64_t>(), indptr.numel(), coo_size);
  return {indices, col};
}

std::pair<torch::Tensor, torch::Tensor> DCSC2COOCUDA(torch::Tensor indptr,
                                                     torch::Tensor indices,
                                                     torch::Tensor ids) {
  auto coo_size = indices.numel();
  auto col = torch::zeros(coo_size, indptr.options());

  dim3 block(128);
  dim3 grid((coo_size + block.x - 1) / block.x);
  _RepeatKernel<int64_t, true>
      <<<grid, block>>>(indptr.data_ptr<int64_t>(), ids.data_ptr<int64_t>(),
                        col.data_ptr<int64_t>(), indptr.numel(), coo_size);
  return {indices, col};
}

template <typename T>
int _NumberOfBits(const T& range) {
  if (range <= 1) {
    // ranges of 0 or 1 require no bits to store
    return 0;
  }

  int bits = 1;
  while (bits < static_cast<int>(sizeof(T) * 8) && (1 << bits) < range) {
    ++bits;
  }

  CHECK_EQ((range - 1) >> bits, 0);
  CHECK_NE((range - 1) >> (bits - 1), 0);

  return bits;
}

template <typename IdType>
inline std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> COOSort(
    torch::Tensor coo_key, torch::Tensor coo_value, int64_t num_keys) {
  auto num_items = coo_key.numel();
  const int num_bits = _NumberOfBits(num_keys);

  torch::Tensor input_key = coo_key;
  torch::Tensor input_value;
  torch::Tensor output_key = torch::zeros_like(coo_key);
  torch::Tensor output_value;

  input_value = torch::arange(num_items,
                              torch::dtype(torch::kInt64).device(torch::kCUDA));
  output_value = torch::zeros_like(input_value);
  cub_sortPairs<IdType, int64_t>(
      input_key.data_ptr<IdType>(), output_key.data_ptr<IdType>(),
      input_value.data_ptr<int64_t>(), output_value.data_ptr<int64_t>(),
      num_items, num_bits);

  return {output_key, coo_value.index({output_value}), output_value};
}

/*!
 * \brief Search for the insertion positions for needle in the hay.
 *
 * The hay is a list of sorted elements and the result is the insertion position
 * of each needle so that the insertion still gives sorted order.
 *
 * It essentially perform binary search to find upper bound for each needle
 * elements.
 *
 * For example:
 * hay = [0, 0, 1, 2, 2]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 5, 5]
 *
 * hay = [0, 0, 1, 3, 3]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 3, 5]
 */
template <typename IdType>
__global__ void _SortedSearchKernelUpperBound(const IdType* hay,
                                              int64_t hay_size,
                                              int64_t num_needles,
                                              IdType* pos) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    pos[tx] = hipcub::UpperBound(hay, hay_size, tx);
    tx += stride_x;
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::optional<torch::Tensor>>
COO2CSCCUDA(torch::Tensor row, torch::Tensor col, int64_t num_cols,
            bool col_sorted) {
  torch::Tensor sort_row, sort_col;
  torch::optional<torch::Tensor> sort_index;
  if (col_sorted) {
    sort_col = col, sort_row = row;
    sort_index = torch::nullopt;
  } else {
    std::tie(sort_col, sort_row, sort_index) =
        COOSort<int64_t>(col, row, num_cols);
  }

  auto indptr = torch::zeros(num_cols + 1, sort_col.options());

  dim3 block(128);
  dim3 grid((num_cols + block.x - 1) / block.x);
  _SortedSearchKernelUpperBound<int64_t>
      <<<grid, block>>>(sort_col.data_ptr<int64_t>(), sort_col.numel(),
                        num_cols, indptr.data_ptr<int64_t>() + 1);
  return std::make_tuple(indptr, sort_row, sort_index);
}

template <typename IdType>
__global__ void _SortedSearchKernelUpperBoundWithMapping(const IdType* hay,
                                                         int64_t hay_size,
                                                         int64_t num_needles,
                                                         IdType* pos,
                                                         IdType* map) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    pos[tx] = hipcub::UpperBound(hay, hay_size, map[tx]);
    tx += stride_x;
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::optional<torch::Tensor>,
           torch::Tensor>
COO2DCSCCUDA(torch::Tensor row, torch::Tensor col, int64_t max_num_cols,
             bool col_sorted) {
  torch::Tensor sort_row, sort_col;
  torch::optional<torch::Tensor> sort_index;
  if (col_sorted) {
    sort_col = col, sort_row = row;
    sort_index = torch::nullopt;
  } else {
    std::tie(sort_col, sort_row, sort_index) =
        COOSort<int64_t>(col, row, max_num_cols);
  }

  auto d_unique_res = torch::empty_like(col);
  auto d_num_selected_out = torch::empty(1, col.options());
  cub_consecutiveUnique<int64_t>(
      sort_col.data_ptr<int64_t>(), d_unique_res.data_ptr<int64_t>(),
      d_num_selected_out.data_ptr<int64_t>(), col.numel());
  thrust::device_ptr<int64_t> item_prefix(
      static_cast<int64_t*>(d_num_selected_out.data_ptr<int64_t>()));
  auto val_col_ids = d_unique_res.index(
      {torch::indexing::Slice(torch::indexing::None, item_prefix[0])});

  auto id_size = val_col_ids.numel();
  auto indptr = torch::zeros(id_size + 1, sort_row.options());

  dim3 block(128);
  dim3 grid((id_size + block.x - 1) / block.x);
  _SortedSearchKernelUpperBoundWithMapping<int64_t><<<grid, block>>>(
      sort_col.data_ptr<int64_t>(), sort_col.numel(), id_size,
      indptr.data_ptr<int64_t>() + 1, val_col_ids.data_ptr<int64_t>());
  return std::make_tuple(indptr, sort_row, sort_index, val_col_ids);
}

}  // namespace impl
}  // namespace gs
