#include "hip/hip_runtime.h"
#include "graph_ops.h"

#include "cuda_common.h"
#include "utils.h"

namespace gs {
namespace impl {

/*!
 * @brief Repeat elements.
 *
 * @param pos: The position of the output buffer to write the value.
 * @param out: Output buffer.
 * @param n_col: Length of positions
 * @param length: Number of values
 *
 * For example:
 * pos = [0, 1, 3, 4]
 * (implicit) val = [0, 1, 2]
 * then,
 * out = [0, 1, 1, 2]
 */
template <typename IdType>
__global__ void _RepeatKernel(const IdType* pos, IdType* out, int64_t n_col,
                              int64_t length) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    IdType i = hipcub::UpperBound(pos, n_col, tx) - 1;
    out[tx] = i;
    tx += stride_x;
  }
}

std::pair<torch::Tensor, torch::Tensor> CSC2COOCUDA(
    torch::Tensor indptr, torch::Tensor indices) {
  auto coo_size = indices.numel();
  auto col = torch::zeros(coo_size, indptr.options());

  dim3 block(128);
  dim3 grid((coo_size + block.x - 1) / block.x);
  _RepeatKernel<int64_t><<<grid, block>>>(indptr.data_ptr<int64_t>(),
                                          col.data_ptr<int64_t>(),
                                          indptr.numel(), coo_size);
  return {indices, col};
}

template <typename IdType>
inline std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> COOSort(
    torch::Tensor coo_key, torch::Tensor coo_value) {
  int num_items = coo_key.numel();

  torch::Tensor input_key = coo_key;
  torch::Tensor input_value;
  torch::Tensor output_key = torch::zeros_like(coo_key);
  torch::Tensor output_value;

  input_value = torch::arange(num_items,
                              torch::dtype(torch::kInt64).device(torch::kCUDA));
  output_value = torch::zeros_like(input_value);
  cub_sortPairs<IdType, int64_t>(input_key.data_ptr<IdType>(),
                                 output_key.data_ptr<IdType>(),
                                 input_value.data_ptr<int64_t>(),
                                 output_value.data_ptr<int64_t>(), num_items);

  return {output_key, coo_value.index({output_value}), output_value};
}

/*!
 * \brief Search for the insertion positions for needle in the hay.
 *
 * The hay is a list of sorted elements and the result is the insertion position
 * of each needle so that the insertion still gives sorted order.
 *
 * It essentially perform binary search to find upper bound for each needle
 * elements.
 *
 * For example:
 * hay = [0, 0, 1, 2, 2]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 5, 5]
 *
 * hay = [0, 0, 1, 3, 3]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 3, 5]
 */
template <typename IdType>
__global__ void _SortedSearchKernelUpperBound(const IdType* hay,
                                              int64_t hay_size,
                                              int64_t num_needles,
                                              IdType* pos) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    pos[tx] = hipcub::UpperBound(hay, hay_size, tx);
    tx += stride_x;
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> COO2CSCCUDA(
    torch::Tensor row, torch::Tensor col, int64_t num_rows) {
  torch::Tensor sort_row, sort_col, sort_index;
  std::tie(sort_row, sort_col, sort_index) = COOSort<int64_t>(col, row);

  auto row_size = num_rows;
  auto indptr = torch::zeros(row_size + 1, sort_row.options());

  dim3 block(128);
  dim3 grid((row_size + block.x - 1) / block.x);
  _SortedSearchKernelUpperBound<int64_t>
      <<<grid, block>>>(sort_row.data_ptr<int64_t>(), sort_row.numel(),
                        row_size, indptr.data_ptr<int64_t>() + 1);
  return std::make_tuple(indptr, sort_col, sort_index);
}

}  // namespace impl
}  // namespace gs
