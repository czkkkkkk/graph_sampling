#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "atomic.h"
#include "cuda_common.h"
#include "tensor_ops.h"
#include "utils.h"

namespace gs {
namespace impl {

template <typename IdType, int64_t BLOCK_WARPS>
__global__ void _SplitIndptrBySize(IdType* indptr, IdType* output, int64_t size,
                                   int64_t num_batchs) {
  assert(blockDim.x == 32);
  assert(blockDim.y == BLOCK_WARPS);

  int laneid = threadIdx.x;
  int warp_id = blockIdx.x * blockDim.y + threadIdx.y;

  for (int i = warp_id; i < num_batchs; i += gridDim.x * blockDim.y) {
    int64_t indptr_begin = size * i;
    int64_t out_begin = (size + 1) * i;
    int64_t offset = indptr[indptr_begin];
    for (int j = laneid; j < size + 1; j += WARP_SIZE) {
      output[j + out_begin] = indptr[j + indptr_begin] - offset;
    }
  }
}

template <typename IdType>
std::vector<torch::Tensor> SplitIndptrBySize(torch::Tensor indptr,
                                             int64_t size) {
  int64_t num_batchs = (indptr.numel() - 1) / size;

  torch::Tensor split_indptr =
      torch::empty(num_batchs * (size + 1), indptr.options());

  constexpr int64_t BLOCK_WARPS = 4;
  dim3 block(WARP_SIZE, BLOCK_WARPS);
  dim3 grid((num_batchs + BLOCK_WARPS - 1) / BLOCK_WARPS);

  _SplitIndptrBySize<IdType, BLOCK_WARPS>
      <<<grid, block>>>(indptr.data_ptr<IdType>(),
                        split_indptr.data_ptr<IdType>(), size, num_batchs);

  return torch::split(split_indptr, (size + 1));
}

template <typename IdType, int64_t BLOCK_WARPS>
__global__ void _SplitIndptrByOffsetKernel(IdType* indptr, IdType* output,
                                           int64_t* offsets,
                                           int64_t num_batchs) {
  assert(blockDim.x == 32);
  assert(blockDim.y == BLOCK_WARPS);

  int laneid = threadIdx.x;
  int warp_id = blockIdx.x * blockDim.y + threadIdx.y;

  for (int i = warp_id; i < num_batchs; i += gridDim.x * blockDim.y) {
    int64_t in_begin = offsets[i];
    int64_t in_end = offsets[i + 1];
    int64_t out_begin = in_begin + warp_id;
    int64_t prefix = indptr[in_begin];
    for (int j = laneid; j < in_end - in_begin + 1; j += WARP_SIZE) {
      output[j + out_begin] = indptr[j + in_begin] - prefix;
    }
  }
}

template <typename IdType>
std::vector<torch::Tensor> SplitIndptrByOffset(torch::Tensor indptr,
                                               torch::Tensor offsets) {
  int64_t num_batchs = offsets.numel() - 1;
  torch::Tensor size_tensor =
      offsets.slice(0, 1, num_batchs + 1) - offsets.slice(0, 0, num_batchs) + 1;
  int64_t total_len = torch::sum(size_tensor).item<int64_t>();

  torch::Tensor split_indptr = torch::empty(total_len, indptr.options());

  constexpr int64_t BLOCK_WARPS = 4;
  dim3 block(WARP_SIZE, BLOCK_WARPS);
  dim3 grid((num_batchs + BLOCK_WARPS - 1) / BLOCK_WARPS);

  CUDA_KERNEL_CALL((_SplitIndptrByOffsetKernel<IdType, BLOCK_WARPS>), grid,
                   block, indptr.data_ptr<IdType>(),
                   split_indptr.data_ptr<IdType>(), offsets.data_ptr<int64_t>(),
                   num_batchs);

  size_tensor = size_tensor.to(torch::kCPU);
  auto data_ptr = size_tensor.data_ptr<int64_t>();
  std::vector<int64_t> split(data_ptr, data_ptr + size_tensor.numel());
  at::IntArrayRef split_sizes(split.data(), split.size()); 
  return torch::split_with_sizes(split_indptr, split_sizes);
}

std::vector<torch::Tensor> SplitIndptrBySizeCUDA(torch::Tensor indptr,
                                                 int64_t size) {
  return SplitIndptrBySize<int64_t>(indptr, size);
}

std::vector<torch::Tensor> SplitIndptrByOffsetCUDA(torch::Tensor indptr,
                                                   torch::Tensor offsets) {
  if(indptr.scalar_type()==torch::kInt64){
 //   std::cout<<__FILE__<<":"<<__LINE__<<std::endl;
  return SplitIndptrByOffset<int64_t>(indptr, offsets);
}
else{
// std::cout<<__FILE__<<__LINE__<<std::endl;
  return SplitIndptrByOffset<int32_t>(indptr, offsets);
}

}
}  // namespace impl
}  // namespace gs