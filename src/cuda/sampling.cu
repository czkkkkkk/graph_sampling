#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "atomic.h"
#include "cuda_common.h"
#include "graph_ops.h"
#include "utils.h"

namespace gs {
namespace impl {

/////////////////////////// CSCColSamplingCUDA /////////////////////////////////
template <typename IdType, bool WITH_COO>
__global__ void _SampleSubIndicesReplaceKernel(IdType* sub_indices,
                                               IdType* select_index,
                                               IdType* coo_row, IdType* indptr,
                                               IdType* indices,
                                               IdType* sub_indptr, int64_t size,
                                               const uint64_t random_seed) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (row < size) {
    IdType in_start = indptr[row];
    IdType out_start = sub_indptr[row];
    IdType degree = indptr[row + 1] - in_start;
    IdType fanout = sub_indptr[row + 1] - out_start;
    IdType out_pos, in_pos;
    for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
      const IdType edge = hiprand(&rng) % degree;
      out_pos = out_start + idx;
      in_pos = in_start + edge;
      sub_indices[out_pos] = indices[in_pos];
      select_index[out_pos] = in_pos;
      if (WITH_COO) {
        coo_row[out_pos] = row;
      }
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType, bool WITH_COO>
__global__ void _SampleSubIndicesKernel(IdType* sub_indices,
                                        IdType* select_index, IdType* coo_row,
                                        IdType* indptr, IdType* indices,
                                        IdType* sub_indptr, int64_t size,
                                        const uint64_t random_seed) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (row < size) {
    IdType in_start = indptr[row];
    IdType out_start = sub_indptr[row];
    IdType degree = indptr[row + 1] - in_start;
    IdType fanout = sub_indptr[row + 1] - out_start;
    IdType out_pos, in_pos;
    if (degree <= fanout) {
      for (int idx = threadIdx.x; idx < degree; idx += blockDim.x) {
        out_pos = out_start + idx;
        in_pos = in_start + idx;
        sub_indices[out_pos] = indices[in_pos];
        select_index[out_pos] = in_pos;
        if (WITH_COO) {
          coo_row[out_pos] = row;
        }
      }
    } else {
      // reservoir algorithm
      for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
        sub_indices[out_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = fanout + threadIdx.x; idx < degree; idx += blockDim.x) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < fanout) {
          AtomicMax(sub_indices + out_start + num, IdType(idx));
        }
      }
      __syncthreads();

      for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
        out_pos = out_start + idx;
        const IdType perm_idx = in_start + sub_indices[out_pos];
        sub_indices[out_pos] = indices[perm_idx];
        select_index[out_pos] = perm_idx;
        if (WITH_COO) {
          coo_row[out_pos] = row;
        }
      }
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType, bool WITH_COO>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
_CSCColSampling(torch::Tensor indptr, torch::Tensor indices, int64_t fanout,
                bool replace) {
  int64_t num_items = indptr.numel() - 1;
  auto sub_indptr = torch::empty(num_items + 1, indptr.options());

  // compute indptr
  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(thrust::device, it(0), it(num_items),
                   [in_indptr = indptr.data_ptr<IdType>(),
                    out = sub_indptr.data_ptr<IdType>(), if_replace = replace,
                    num_fanout = fanout] __device__(int i) mutable {
                     IdType begin = in_indptr[i];
                     IdType end = in_indptr[i + 1];
                     if (if_replace) {
                       out[i] = (end - begin) == 0 ? 0 : num_fanout;
                     } else {
                       out[i] = min(end - begin, num_fanout);
                     }
                   });
  cub_exclusiveSum<IdType>(sub_indptr.data_ptr<IdType>(), num_items + 1);

  // compute indices
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int nnz = item_prefix[num_items];  // cpu
  auto sub_indices = torch::empty(nnz, indices.options());
  auto select_index = torch::empty(nnz, indices.options());

  torch::Tensor coo_col;
  IdType* coo_col_ptr;
  if (WITH_COO) {
    coo_col = torch::empty(nnz, indices.options());
    coo_col_ptr = coo_col.data_ptr<IdType>();
  } else {
    coo_col = torch::Tensor();
    coo_col_ptr = nullptr;
  }

  const uint64_t random_seed = 7777;
  dim3 block(32, 16);
  dim3 grid((num_items + block.x - 1) / block.x);
  if (replace) {
    _SampleSubIndicesReplaceKernel<IdType, WITH_COO><<<grid, block>>>(
        sub_indices.data_ptr<IdType>(), select_index.data_ptr<IdType>(),
        coo_col_ptr, indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
        sub_indptr.data_ptr<IdType>(), num_items, random_seed);
  } else {
    _SampleSubIndicesKernel<IdType, WITH_COO><<<grid, block>>>(
        sub_indices.data_ptr<IdType>(), select_index.data_ptr<IdType>(),
        coo_col_ptr, indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
        sub_indptr.data_ptr<IdType>(), num_items, random_seed);
  }

  return {sub_indptr, coo_col, sub_indices, select_index};
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
CSCColSamplingCUDA(torch::Tensor indptr, torch::Tensor indices, int64_t fanout,
                   bool replace, bool with_coo) {
  torch::Tensor out_indptr, out_coo_col, out_indices, out_selected_index;
  if (with_coo)
    std::tie(out_indptr, out_coo_col, out_indices, out_selected_index) =
        _CSCColSampling<int64_t, true>(indptr, indices, fanout, replace);
  else
    std::tie(out_indptr, out_coo_col, out_indices, out_selected_index) =
        _CSCColSampling<int64_t, false>(indptr, indices, fanout, replace);
  return {out_indptr, out_coo_col, out_indices, out_selected_index};
}

}  // namespace impl
}  // namespace gs