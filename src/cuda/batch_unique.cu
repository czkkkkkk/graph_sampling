#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "atomic.h"
#include "cuda_common.h"
#include "tensor_ops.h"
#include "utils.h"

namespace gs {
namespace impl {

template <typename IdType>
struct RelabelHashmap {
  __device__ inline RelabelHashmap(IdType* Kptr, IdType* Vptr, size_t numel)
      : kptr(Kptr), vptr(Vptr), capacity(numel){};

  __device__ inline void Update(IdType key, IdType value) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);
    IdType prev = AtomicCAS(&kptr[pos], kEmptyKey, key);

    while (prev != key and prev != kEmptyKey) {
      pos = hash(pos + delta);
      delta += 1;
      prev = AtomicCAS(&kptr[pos], kEmptyKey, key);
    }

    AtomicMin(vptr + pos, value);
  }

  __device__ inline IdType SearchForPos(IdType key) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);

    while (true) {
      if (kptr[pos] == key) {
        return pos;
      }
      if (kptr[pos] == kEmptyKey) {
        return -1;
      }
      pos = hash(pos + delta);
      delta += 1;
    }
  }

  __device__ inline IdType SearchForValue(IdType key) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);

    while (true) {
      if (kptr[pos] == key) {
        return vptr[pos];
      };
      if (kptr[pos] == kEmptyKey) {
        return -1;
      }
      pos = hash(pos + delta);
      delta += 1;
    }
  }

  __device__ inline uint32_t hash(int32_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(uint32_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(int64_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(uint64_t key) { return key & (capacity - 1); }

  IdType kEmptyKey{-1};
  IdType* kptr;
  IdType* vptr;
  uint32_t capacity{0};
};

template <typename IdType>
__global__ void _InsertHashmaps(IdType* __restrict__ data_tensor,
                                IdType* __restrict__ data_key_tensor,
                                IdType* __restrict__ hashmap_key_tensor,
                                IdType* __restrict__ hashmap_value_tensor,
                                IdType* __restrict__ hashmap_ptr,
                                int64_t num_items) {
  int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int64_t index = tid; index < num_items;
       index += gridDim.x * blockDim.x) {
    int64_t batch_index = data_key_tensor[index];
    int64_t hashmap_begin = hashmap_ptr[batch_index];
    int64_t dir_size = hashmap_ptr[batch_index + 1] - hashmap_begin;
    RelabelHashmap<IdType> table(hashmap_key_tensor + hashmap_begin,
                                 hashmap_value_tensor + hashmap_begin,
                                 dir_size);
    table.Update(data_tensor[index], index);
  }
}

template <typename IdType>
__global__ void _SearchHashmapsForUnique(
    IdType* __restrict__ data_tensor, IdType* __restrict__ data_key_tensor,
    IdType* __restrict__ hashmap_key_tensor,
    IdType* __restrict__ hashmap_value_tensor, IdType* __restrict__ hashmap_ptr,
    IdType* __restrict__ item_prefix_tensor, int64_t num_items) {
  int64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

  for (int64_t index = tid; index < num_items;
       index += gridDim.x * blockDim.x) {
    int64_t batch_index = data_key_tensor[index];
    int64_t hashmap_begin = hashmap_ptr[batch_index];
    int64_t dir_size = hashmap_ptr[batch_index + 1] - hashmap_begin;
    RelabelHashmap<IdType> table(hashmap_key_tensor + hashmap_begin,
                                 hashmap_value_tensor + hashmap_begin,
                                 dir_size);
    IdType result = table.SearchForValue(data_tensor[index]);
    item_prefix_tensor[index] = result == index ? 1 : 0;
  }
}

///////////////////////////// BatchUniqueByKey ////////////////////////////////
template <typename IdType>
__global__ void _SortedSearchKernelUpperBound(const IdType* __restrict__ hay,
                                              int64_t hay_size,
                                              int64_t num_needles,
                                              IdType* __restrict__ pos) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    pos[tx] = hipcub::UpperBound(hay, hay_size, tx);
    tx += stride_x;
  }
}

template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> _BatchUniqueByKey(
    torch::Tensor data_tensor, torch::Tensor data_ptr, torch::Tensor data_key) {
  int64_t num_batchs = data_ptr.numel() - 1;
  int64_t num_items = data_tensor.numel();
  torch::Tensor hashmap_ptr = torch::empty_like(data_ptr);

  // Create Hashmaps
  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      it(0), it(num_batchs),
      [in = data_ptr.data_ptr<IdType>(),
       out = hashmap_ptr.data_ptr<IdType>()] __device__(IdType i) mutable {
        out[i] = (1 << static_cast<uint32_t>(log2(in[i + 1] - in[i]) + 1));
      });

  cub_exclusiveSum<IdType>(hashmap_ptr.data_ptr<IdType>(), num_batchs + 1);
  thrust::device_ptr<IdType> wrapper_hashmap_ptr(
      static_cast<IdType*>(hashmap_ptr.data_ptr<IdType>()));
  IdType total_dir_size = wrapper_hashmap_ptr[num_batchs];

  IdType MAX = std::numeric_limits<IdType>::max();
  torch::Tensor key_tensor =
      torch::full(total_dir_size, -1, data_tensor.options());
  torch::Tensor index_tensor =
      torch::full(total_dir_size, MAX, data_tensor.options());

  constexpr int BLOCK_SIZE = 256;
  int num_blocks = (num_items + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 blocks(BLOCK_SIZE);
  dim3 grids(num_blocks);

  _InsertHashmaps<IdType><<<grids, blocks>>>(
      data_tensor.data_ptr<IdType>(), data_key.data_ptr<IdType>(),
      key_tensor.data_ptr<IdType>(), index_tensor.data_ptr<IdType>(),
      hashmap_ptr.data_ptr<IdType>(), num_items);

  torch::Tensor mask_tensor = torch::empty(num_items, data_tensor.options());

  _SearchHashmapsForUnique<IdType><<<grids, blocks>>>(
      data_tensor.data_ptr<IdType>(), data_key.data_ptr<IdType>(),
      key_tensor.data_ptr<IdType>(), index_tensor.data_ptr<IdType>(),
      hashmap_ptr.data_ptr<IdType>(), mask_tensor.data_ptr<IdType>(),
      num_items);

  torch::Tensor data_unique_index = torch::nonzero(mask_tensor).reshape({-1});
  torch::Tensor sub_data = data_tensor.index({data_unique_index});
  torch::Tensor sub_data_key = data_key.index({data_unique_index});

  torch::Tensor sub_data_ptr = torch::zeros_like(data_ptr);

  dim3 block(128);
  dim3 grid((num_batchs + block.x - 1) / block.x);
  _SortedSearchKernelUpperBound<IdType>
      <<<grid, block>>>(sub_data_key.data_ptr<IdType>(), sub_data_key.numel(),
                        num_batchs, sub_data_ptr.data_ptr<IdType>() + 1);

  return {sub_data, sub_data_ptr, sub_data_key};
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> BatchUniqueByKeyCUDA(
    torch::Tensor data_tensor, torch::Tensor data_ptr, torch::Tensor data_key) {
  return _BatchUniqueByKey<int64_t>(data_tensor, data_ptr, data_key);
}

///////////////////////////// BatchUnique ////////////////////////////////
template <typename IdType>
__global__ void _RepeatKernel(const IdType* pos, IdType* out, int64_t n_col,
                              int64_t length) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < length) {
    IdType i = hipcub::UpperBound(pos, n_col, tx) - 1;
    out[tx] = i;
    tx += stride_x;
  }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> BatchUniqueCUDA(
    torch::Tensor data_tensor, torch::Tensor data_ptr) {
  torch::Tensor data_key = torch::empty_like(data_tensor);
  dim3 block(128);
  dim3 grid((data_tensor.numel() + block.x - 1) / block.x);
  _RepeatKernel<int64_t><<<grid, block>>>(
      data_ptr.data_ptr<int64_t>(), data_key.data_ptr<int64_t>(),
      data_ptr.numel(), data_tensor.numel());
  return _BatchUniqueByKey<int64_t>(data_tensor, data_ptr, data_key);
}

///////////////////////////// BatchUnique2 ////////////////////////////////
template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> _BatchUniqueByKey2(
    torch::Tensor data, torch::Tensor data_ptr, torch::Tensor data_key) {
  torch::Tensor segment_sorted_data = torch::empty_like(data);
  int64_t num_segments = data_ptr.numel() - 1;
  int64_t num_items = data.numel();

  cub_segmentSort<IdType>(data.data_ptr<IdType>(),
                          segment_sorted_data.data_ptr<IdType>(),
                          data_ptr.data_ptr<IdType>(), num_items, num_segments);

  torch::Tensor unique_data = torch::empty_like(segment_sorted_data);
  torch::Tensor unique_data_key = torch::empty_like(data_key);

  int64_t* d_num_selected_out =
      c10::cuda::CUDACachingAllocator::raw_alloc(sizeof(int64_t));

  cub_uniqueByKey<IdType, IdType>(
      segment_sorted_data.data_ptr<IdType>(), unique_data.data_ptr<IdType>(),
      data_key.data_ptr<IdType>(), unique_data_key.data_ptr<IdType>(),
      num_items, d_num_selected_out);

  thrust::device_ptr<int64_t> wrapper_d_num_selected_out(d_num_selected_out);
  int64_t num_unique = wrapper_d_num_selected_out[0];

  torch::Tensor narrow_unique_tensor = unique_data.slice(num_unique);
  torch::Tensor narrow_unique_tensor_key = unique_data_key.slice(num_unique);

  torch::Tensor narrow_unique_tensor_ptr = torch::zeros_like(data_ptr);

  dim3 block(128);
  dim3 grid((num_segments + block.x - 1) / block.x);
  _SortedSearchKernelUpperBound<IdType>
      <<<grid, block>>>(narrow_unique_tensor_key.data_ptr<IdType>(),
                        narrow_unique_tensor_key.numel(), num_segments,
                        narrow_unique_tensor_ptr.data_ptr<IdType>() + 1);

  return {narrow_unique_tensor, narrow_unique_tensor_ptr,
          narrow_unique_tensor_key};
}

}  // namespace impl
}  // namespace gs