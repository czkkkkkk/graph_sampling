#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "atomic.h"
#include "cuda_common.h"
#include "graph_ops.h"
#include "utils.h"

namespace gs {
namespace impl {

/////////////////////////// CSCColSamplingCUDA /////////////////////////////////
template <typename IdType>
__global__ void _SampleSubIndicesReplaceKernel(IdType* coo_row,
                                               IdType* select_index,
                                               IdType* coo_col, IdType* indptr,
                                               IdType* indices,
                                               IdType* sub_indptr, int64_t size,
                                               const uint64_t random_seed) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (row < size) {
    IdType in_start = indptr[row];
    IdType out_start = sub_indptr[row];
    IdType degree = indptr[row + 1] - in_start;
    IdType fanout = sub_indptr[row + 1] - out_start;
    IdType out_pos, in_pos;
    for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
      const IdType edge = hiprand(&rng) % degree;
      out_pos = out_start + idx;
      in_pos = in_start + edge;
      select_index[out_pos] = in_pos;
      coo_row[out_pos] = indices[in_pos];
      coo_col[out_pos] = row;
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
__global__ void _SampleSubIndicesKernel(IdType* coo_row, IdType* select_index,
                                        IdType* coo_col, IdType* indptr,
                                        IdType* indices, IdType* sub_indptr,
                                        int64_t size,
                                        const uint64_t random_seed) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (row < size) {
    IdType in_start = indptr[row];
    IdType out_start = sub_indptr[row];
    IdType degree = indptr[row + 1] - in_start;
    IdType fanout = sub_indptr[row + 1] - out_start;
    IdType out_pos, in_pos;
    if (degree <= fanout) {
      for (int idx = threadIdx.x; idx < degree; idx += blockDim.x) {
        out_pos = out_start + idx;
        in_pos = in_start + idx;

        select_index[out_pos] = in_pos;
        coo_row[out_pos] = indices[in_pos];
        coo_col[out_pos] = row;
      }
    } else {
      // reservoir algorithm
      for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
        coo_row[out_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = fanout + threadIdx.x; idx < degree; idx += blockDim.x) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < fanout) {
          AtomicMax(coo_row + out_start + num, IdType(idx));
        }
      }
      __syncthreads();

      for (int idx = threadIdx.x; idx < fanout; idx += blockDim.x) {
        out_pos = out_start + idx;
        const IdType perm_idx = in_start + coo_row[out_pos];
        coo_row[out_pos] = indices[perm_idx];
        select_index[out_pos] = perm_idx;
        coo_col[out_pos] = row;
      }
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> __CSCColSampling(
    torch::Tensor indptr, torch::Tensor indices, int64_t fanout, bool replace) {
  int64_t num_items = indptr.numel() - 1;
  auto sub_indptr = torch::empty(
      num_items + 1, torch::dtype(indptr.dtype()).device(torch::kCUDA));

  // compute indptr
  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(thrust::device, it(0), it(num_items),
                   [in_indptr = indptr.data_ptr<IdType>(),
                    out = sub_indptr.data_ptr<IdType>(), if_replace = replace,
                    num_fanout = fanout] __device__(int i) mutable {
                     IdType begin = in_indptr[i];
                     IdType end = in_indptr[i + 1];
                     if (if_replace) {
                       out[i] = (end - begin) == 0 ? 0 : num_fanout;
                     } else {
                       out[i] = min(end - begin, num_fanout);
                     }
                   });
  cub_exclusiveSum<IdType>(sub_indptr.data_ptr<IdType>(), num_items + 1);

  // compute indices
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int nnz = item_prefix[num_items];  // cpu
  auto coo_row =
      torch::ones(nnz, torch::dtype(indices.dtype()).device(torch::kCUDA));
  auto coo_col =
      torch::empty(nnz, torch::dtype(indices.dtype()).device(torch::kCUDA));
  auto select_index =
      torch::empty(nnz, torch::dtype(indices.dtype()).device(torch::kCUDA));

  const uint64_t random_seed = 7777;
  dim3 block(32, 16);
  dim3 grid((num_items + block.x - 1) / block.x);
  if (replace) {
    _SampleSubIndicesReplaceKernel<IdType><<<grid, block>>>(
        coo_row.data_ptr<IdType>(), select_index.data_ptr<IdType>(),
        coo_col.data_ptr<IdType>(), indptr.data_ptr<IdType>(),
        indices.data_ptr<IdType>(), sub_indptr.data_ptr<IdType>(), num_items,
        random_seed);
  } else {
    _SampleSubIndicesKernel<IdType><<<grid, block>>>(
        coo_row.data_ptr<IdType>(), select_index.data_ptr<IdType>(),
        coo_col.data_ptr<IdType>(), indptr.data_ptr<IdType>(),
        indices.data_ptr<IdType>(), sub_indptr.data_ptr<IdType>(), num_items,
        random_seed);
  }

  return {coo_row, coo_col, select_index};
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> FullCSCColSamplingCUDA(
    torch::Tensor indptr, torch::Tensor indices, int64_t fanout, bool replace) {
  torch::Tensor coo_row, coo_col, selected_index;
  std::tie(coo_row, coo_col, selected_index) =
      __CSCColSampling<int64_t>(indptr, indices, fanout, replace);
  return {coo_row, coo_col, selected_index};
}

}  // namespace impl
}  // namespace gs