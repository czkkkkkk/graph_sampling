#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "../atomic.h"
#include "../cuda_common.h"
#include "../utils.h"

#include "batch_ops.h"

namespace gs {
namespace impl {
namespace batch {
template <typename IdType>
__global__ void _BatchTensorEncodingKernel(IdType* out_data, IdType* in_data,
                                           int64_t* in_offsets,
                                           int64_t num_batch,
                                           int64_t seg_size) {
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  const int stride_y = blockDim.y * gridDim.y;
  while (ty < num_batch) {
    int64_t in_startoff = in_offsets[ty];
    int64_t in_endoff = in_offsets[ty + 1];
    int size = in_endoff - in_startoff;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride_x = blockDim.x * gridDim.x;
    for (; tx < size; tx += stride_x) {
      out_data[in_startoff + tx] = ty * seg_size + in_data[in_startoff + tx];
    }
    ty += stride_y;
  }
}

/*!
 * \brief Search for the insertion positions for needle in the hay.
 *
 * The hay is a list of sorted elements and the result is the insertion position
 * of each needle so that the insertion still gives sorted order.
 *
 * It essentially perform binary search to find upper bound for each needle
 * elements.
 *
 * For example:
 * hay = [0, 0, 1, 2, 2]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 5, 5]
 *
 * hay = [0, 0, 1, 3, 3]
 * (implicit) needle = [0, 1, 2, 3]
 * then,
 * out = [2, 3, 3, 5]
 */
template <typename IdType>
__global__ void _SortedSearchKernelUpperBound(const IdType* hay,
                                              int64_t hay_size,
                                              int64_t num_needles,
                                              IdType* pos) {
  IdType tx = static_cast<IdType>(blockIdx.x) * blockDim.x + threadIdx.x;
  const int stride_x = gridDim.x * blockDim.x;
  while (tx < num_needles) {
    pos[tx] = hipcub::UpperBound(hay, hay_size, tx);
    tx += stride_x;
  }
}

torch::Tensor BatchEncodeCUDA(torch::Tensor data_tensor, torch::Tensor data_ptr,
                              int64_t encoding_size) {
  torch::Tensor out_data = torch::empty_like(data_tensor);
  int64_t numel = data_ptr.numel(), num_split = data_ptr.numel() - 1;
  torch::Tensor data_sizes =
      data_ptr.slice(0, 1, numel) - data_ptr.slice(0, 0, numel - 1);
  int64_t max_len = data_sizes.max().item<int64_t>();

  const int ntx = FindNumThreads(max_len);
  const int nty = CUDA_MAX_NUM_THREADS / ntx;
  const int nbx = (max_len + ntx - 1) / ntx;
  const int nby = FindNumBlocks<'y'>((num_split + nty - 1) / nty);
  dim3 grid(nbx, nby);
  dim3 block(ntx, nty);
  CUDA_KERNEL_CALL((_BatchTensorEncodingKernel<int64_t>), grid, block,
                   out_data.data_ptr<int64_t>(),
                   data_tensor.data_ptr<int64_t>(),
                   data_ptr.data_ptr<int64_t>(), num_split, encoding_size);
  return out_data;
}

torch::Tensor BatchDecodeCUDA(torch::Tensor in_data, int64_t encoding_size) {
  torch::Tensor out_data = torch::empty_like(in_data);

  using it = thrust::counting_iterator<int64_t>;
  thrust::for_each(
      it(0), it(in_data.numel()),
      [in = in_data.data_ptr<int64_t>(), out = out_data.data_ptr<int64_t>(),
       size = encoding_size] __device__(int64_t i) mutable {
        out[i] = in[i] - (in[i] / size) * size;
      });
  return out_data;
}

std::tuple<torch::Tensor, torch::Tensor> GetBatchOffsets(
    torch::Tensor data_tensor, int64_t num_batches, int64_t encoding_size) {
  auto data_mask = torch::empty_like(data_tensor);
  auto decoded_data = torch::empty_like(data_tensor);
  using it = thrust::counting_iterator<int64_t>;
  thrust::for_each(it(0), it(data_tensor.numel()),
                   [in = data_tensor.data_ptr<int64_t>(),
                    out = decoded_data.data_ptr<int64_t>(),
                    mask = data_mask.data_ptr<int64_t>(),
                    size = encoding_size] __device__(int64_t i) mutable {
                     mask[i] = in[i] / size;
                     out[i] = in[i] - mask[i] * size;
                   });

  auto dataptr = torch::zeros(num_batches + 1,
                              torch::dtype(torch::kInt64).device(torch::kCUDA));

  dim3 block(128);
  dim3 grid((num_batches + block.x - 1) / block.x);
  CUDA_KERNEL_CALL((_SortedSearchKernelUpperBound<int64_t>), grid, block,
                   data_mask.data_ptr<int64_t>(), data_mask.numel(),
                   num_batches, dataptr.data_ptr<int64_t>() + 1);
  return {dataptr, decoded_data};
}
}  // namespace batch
}  // namespace impl
}  // namespace gs