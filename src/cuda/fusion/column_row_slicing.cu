#include "hip/hip_runtime.h"
#include "../atomic.h"
#include "../cuda_common.h"
#include "../utils.h"
#include "column_row_slicing.h"

namespace gs {
namespace impl {
namespace fusion {

inline __host__ __device__ int UpPower(int key) {
  int ret = 1 << static_cast<uint32_t>(std::log2(key) + 1);
  return ret;
}

__device__ inline uint32_t Hash32Shift(uint32_t key) {
  key = ~key + (key << 15);  // # key = (key << 15) - key - 1;
  key = key ^ (key >> 12);
  key = key + (key << 2);
  key = key ^ (key >> 4);
  key = key * 2057;  // key = (key + (key << 3)) + (key << 11);
  key = key ^ (key >> 16);
  return key;
}

__device__ inline uint64_t Hash64Shift(uint64_t key) {
  key = (~key) + (key << 21);  // key = (key << 21) - key - 1;
  key = key ^ (key >> 24);
  key = (key + (key << 3)) + (key << 8);  // key * 265
  key = key ^ (key >> 14);
  key = (key + (key << 2)) + (key << 4);  // key * 21
  key = key ^ (key >> 28);
  key = key + (key << 31);
  return key;
}

/**
 * @brief Used to judge whether a node is in a node set
 *
 * @tparam IdType
 */
template <typename IdType>
struct NodeQueryHashmap {
  __device__ inline NodeQueryHashmap(IdType* Kptr, IdType* Vptr, size_t numel)
      : kptr(Kptr), vptr(Vptr), capacity(numel){};

  __device__ inline void Insert(IdType key, IdType value) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);
    IdType prev = AtomicCAS(&kptr[pos], kEmptyKey, key);

    while (prev != key and prev != kEmptyKey) {
      pos = hash(pos + delta);
      delta += 1;
      prev = AtomicCAS(&kptr[pos], kEmptyKey, key);
    }

    vptr[pos] = value;
  }

  __device__ inline IdType Query(IdType key) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);

    while (true) {
      if (kptr[pos] == key) {
        return vptr[pos];
      }
      if (kptr[pos] == kEmptyKey) {
        return -1;
      }
      pos = hash(pos + delta);
      delta += 1;
    }

    return -1;
  }

  __device__ inline uint32_t hash(int32_t key) {
    return Hash32Shift(key) & (capacity - 1);
  }

  __device__ inline uint32_t hash(uint32_t key) {
    return Hash32Shift(key) & (capacity - 1);
  }

  __device__ inline uint32_t hash(int64_t key) {
    return static_cast<uint32_t>(Hash64Shift(key)) & (capacity - 1);
  }

  __device__ inline uint32_t hash(uint64_t key) {
    return static_cast<uint32_t>(Hash64Shift(key)) & (capacity - 1);
  }

  IdType kEmptyKey{-1};
  IdType* kptr;
  IdType* vptr;
  uint32_t capacity{0};
};

template <typename IdType, int BLOCK_WARPS, int TILE_SIZE>
__global__ void _RowColSlicingKernel(IdType* in_indptr, IdType* in_indices,
                                     IdType* sub_indptr, IdType* seeds,
                                     IdType* key_buffer, IdType* value_buffer,
                                     int num_items, int dir_size,
                                     IdType* out_count, IdType* out_indices,
                                     IdType* out_mask) {
  assert(blockDim.x == WARP_SIZE);
  assert(blockDim.y == BLOCK_WARPS);

  IdType out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
  IdType last_row =
      MIN(static_cast<IdType>(blockIdx.x + 1) * TILE_SIZE, num_items);

  int warp_id = threadIdx.y;
  int laneid = threadIdx.x;

  NodeQueryHashmap<IdType> hashmap(key_buffer, value_buffer, dir_size);

  typedef hipcub::WarpReduce<IdType> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage[BLOCK_WARPS];
  while (out_row < last_row) {
    IdType count = 0;
    IdType row = seeds[out_row];
    IdType in_row_start = in_indptr[row];
    IdType deg = in_indptr[row + 1] - in_row_start;
    IdType out_row_start = sub_indptr[out_row];

    for (int idx = laneid; idx < deg; idx += WARP_SIZE) {
      IdType value = hashmap.Query(in_indices[in_row_start + idx]);
      if (value != -1) {
        count += 1;
        out_mask[out_row_start + idx] = 1;
        out_indices[out_row_start + idx] = value;
      }
    }

    int out_deg = WarpReduce(temp_storage[warp_id]).Sum(count);
    if (laneid == 0) {
      out_count[out_row] = out_deg;
    }

    out_row += BLOCK_WARPS;
  }
}

template <typename IdType>
torch::Tensor GetSubIndptr(torch::Tensor indptr, torch::Tensor seeds) {
  int64_t size = seeds.numel();
  auto new_indptr = torch::zeros(size + 1, indptr.options());

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(size),
      [in = seeds.data_ptr<IdType>(), in_indptr = indptr.data_ptr<IdType>(),
       out = new_indptr.data_ptr<IdType>()] __device__(int i) mutable {
        IdType begin = in_indptr[in[i]];
        IdType end = in_indptr[in[i] + 1];
        out[i] = end - begin;
      });

  cub_exclusiveSum<IdType>(new_indptr.data_ptr<IdType>(), size + 1);
  return new_indptr;
}

template <typename IdType>
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> _RowColSlicingCUDA(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor seeds) {
  int num_items = seeds.numel();

  // construct NodeQueryHashMap
  int dir_size = UpPower(num_items) * 2;
  torch::Tensor key_buffer = torch::full(dir_size, -1, indptr.options());
  torch::Tensor value_buffer = torch::full(dir_size, -1, indices.options());

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(it(0), it(num_items),
                   [key = seeds.data_ptr<IdType>(),
                    _key_buffer = key_buffer.data_ptr<IdType>(),
                    _value_buffer = value_buffer.data_ptr<IdType>(),
                    dir_size] __device__(IdType i) {
                     NodeQueryHashmap<IdType> hashmap(_key_buffer,
                                                      _value_buffer, dir_size);
                     hashmap.Insert(key[i], i);
                   });

  constexpr int BLOCK_WARP = 128 / WARP_SIZE;
  constexpr int TILE_SIZE = 16;
  const dim3 block(WARP_SIZE, BLOCK_WARP);
  const dim3 grid((num_items + TILE_SIZE - 1) / TILE_SIZE);

  // get sub_indptr
  torch::Tensor sub_indptr = GetSubIndptr<IdType>(indptr, seeds);
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int nnz = item_prefix[num_items];  // cp

  torch::Tensor out_indptr = torch::empty_like(sub_indptr);
  torch::Tensor out_indices = torch::empty(nnz, indices.options());
  torch::Tensor out_mask = torch::zeros(nnz, indices.options());

  // query hashmap to get mask
  _RowColSlicingKernel<IdType, BLOCK_WARP, TILE_SIZE><<<grid, block>>>(
      indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
      sub_indptr.data_ptr<IdType>(), seeds.data_ptr<IdType>(),
      key_buffer.data_ptr<IdType>(), value_buffer.data_ptr<IdType>(), num_items,
      dir_size, out_indptr.data_ptr<IdType>(), out_indices.data_ptr<IdType>(),
      out_mask.data_ptr<IdType>());

  // prefix sum to get out_indptr and out_indices_index
  cub_exclusiveSum<IdType>(out_indptr.data_ptr<IdType>(), num_items + 1);
  torch::Tensor select_index = torch::nonzero(out_mask).reshape({
      -1,
  });

  return {out_indptr, out_indices.index({select_index}), select_index};
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> CSCColRowSlicingCUDA(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor seeds) {
  return _RowColSlicingCUDA<int64_t>(indptr, indices, seeds);
};

}  // namespace fusion

}  // namespace impl
}  // namespace gs