#include "hip/hip_runtime.h"
#include "graph_ops.h"

#include <hiprand/hiprand_kernel.h>
#include <nvToolsExt.h>
#include "cuda_common.h"
#include "utils.h"

namespace gs {
namespace impl {

template <typename IdType>
torch::Tensor GetSubIndptr(torch::Tensor indptr, torch::Tensor column_ids) {
  int64_t size = column_ids.numel();
  auto new_indptr = torch::zeros(size + 1, indptr.options());
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(new_indptr.data_ptr<IdType>()));

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(size),
      [in = column_ids.data_ptr<IdType>(),
       in_indptr = indptr.data_ptr<IdType>(),
       out = thrust::raw_pointer_cast(item_prefix)] __device__(int i) mutable {
        IdType begin = in_indptr[in[i]];
        IdType end = in_indptr[in[i] + 1];
        out[i] = end - begin;
      });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix), size + 1);
  return new_indptr;
}

template <typename IdType>
__global__ void _GetSubIndicesKernel(IdType* out_indices, IdType* indptr,
                                     IdType* indices, IdType* sub_indptr,
                                     IdType* column_ids, int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  while (row < size) {
    int64_t in_start = indptr[column_ids[row]];
    int64_t out_start = sub_indptr[row];
    int64_t n_edges = sub_indptr[row + 1] - sub_indptr[row];
    int64_t tid = threadIdx.x;
    while (tid < n_edges) {
      out_indices[out_start + tid] = indices[in_start + tid];
      tid += blockDim.x;
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
torch::Tensor GetSubIndices(torch::Tensor indptr, torch::Tensor indices,
                            torch::Tensor sub_indptr,
                            torch::Tensor column_ids) {
  int64_t size = column_ids.numel();
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int n_edges = item_prefix[size];  // cpu
  auto sub_indices = torch::zeros(n_edges, indices.options());

  dim3 block(32, 8);
  dim3 grid((size + block.x - 1) / block.x);
  _GetSubIndicesKernel<int64_t><<<grid, block>>>(
      sub_indices.data_ptr<int64_t>(), indptr.data_ptr<int64_t>(),
      indices.data_ptr<int64_t>(), sub_indptr.data_ptr<int64_t>(),
      column_ids.data_ptr<int64_t>(), size);
  return sub_indices;
}

// columwise slicing
std::pair<torch::Tensor, torch::Tensor> CSCColumnwiseSlicingCUDA(
    torch::Tensor indptr, torch::Tensor indices, torch::Tensor column_ids) {
  auto sub_indptr = GetSubIndptr<int64_t>(indptr, column_ids);
  auto sub_indices =
      GetSubIndices<int64_t>(indptr, indices, sub_indptr, column_ids);
  return {sub_indptr, sub_indices};
}

template <typename IdType>
torch::Tensor GetSampledSubIndptr(torch::Tensor indptr, int64_t fanout,
                                  bool replace) {
  int64_t size = indptr.numel();
  auto new_indptr = torch::zeros(size, indptr.options());
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(new_indptr.data_ptr<IdType>()));

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(size),
      [in_indptr = indptr.data_ptr<IdType>(),
       out = thrust::raw_pointer_cast(item_prefix), if_replace = replace,
       num_fanout = fanout] __device__(int i) mutable {
        IdType begin = in_indptr[i];
        IdType end = in_indptr[i + 1];
        if (if_replace) {
          out[i] = (end - begin) == 0 ? 0 : num_fanout;
        } else {
          out[i] = min(end - begin, num_fanout);
        }
      });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix), size + 1);
  return new_indptr;
}

template <typename IdType>
__global__ void _SampleSubIndicesKernel(IdType* sub_indices, IdType* indptr,
                                        IdType* indices, IdType* sub_indptr,
                                        int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  const uint64_t random_seed = 7777777;
  hiprandState rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
  while (row < size) {
    int64_t in_start = indptr[row];
    int64_t out_start = sub_indptr[row];
    int64_t degree = indptr[row + 1] - in_start;
    int64_t fanout = sub_indptr[row + 1] - out_start;
    int64_t tid = threadIdx.x;
    while (tid < fanout) {
      // Sequential Sampling
      // const int64_t edge = tid % degree;
      // Random Sampling
      const int64_t edge = hiprand(&rng) % degree;
      sub_indices[out_start + tid] = indices[in_start + edge];
      tid += blockDim.x;
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
torch::Tensor SampleSubIndices(torch::Tensor indptr, torch::Tensor indices,
                               torch::Tensor sub_indptr) {
  int64_t size = sub_indptr.numel() - 1;
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int n_edges = item_prefix[size];  // cpu
  auto sub_indices = torch::zeros(n_edges, indices.options());

  dim3 block(32, 8);
  dim3 grid((size + block.x - 1) / block.x);
  _SampleSubIndicesKernel<int64_t><<<grid, block>>>(
      sub_indices.data_ptr<int64_t>(), indptr.data_ptr<int64_t>(),
      indices.data_ptr<int64_t>(), sub_indptr.data_ptr<int64_t>(), size);
  return sub_indices;
}

// columnwise sampling
std::pair<torch::Tensor, torch::Tensor> CSCColumnwiseSamplingCUDA(
    torch::Tensor indptr, torch::Tensor indices, int64_t fanout, bool replace) {
  auto sub_indptr = GetSampledSubIndptr<int64_t>(indptr, fanout, replace);
  auto sub_indices = SampleSubIndices<int64_t>(indptr, indices, sub_indptr);
  return {sub_indptr, sub_indices};
}

template <typename IdType>
torch::Tensor GetSampledSubIndptrFused(torch::Tensor indptr,
                                       torch::Tensor column_ids, int64_t fanout,
                                       bool replace) {
  int64_t size = column_ids.numel();
  auto sub_indptr = torch::empty(size + 1, indptr.options());
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      thrust::device, it(0), it(size),
      [in = column_ids.data_ptr<IdType>(),
       in_indptr = indptr.data_ptr<IdType>(),
       out = thrust::raw_pointer_cast(item_prefix), if_replace = replace,
       num_fanout = fanout] __device__(int i) mutable {
        IdType begin = in_indptr[in[i]];
        IdType end = in_indptr[in[i] + 1];
        if (if_replace) {
          out[i] = (end - begin) == 0 ? 0 : num_fanout;
        } else {
          out[i] = min(end - begin, num_fanout);
        }
      });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix), size + 1);
  return sub_indptr;
}

template <typename IdType>
__global__ void _SampleSubIndicesKernelFusedWithReplace(
    IdType* sub_indices, IdType* indptr, IdType* indices, IdType* sub_indptr,
    IdType* column_ids, int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  const uint64_t random_seed = 7777777;
  hiprandState rng;
  hiprand_init(random_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);
  while (row < size) {
    int64_t col = column_ids[row];
    int64_t in_start = indptr[col];
    int64_t out_start = sub_indptr[row];
    int64_t degree = indptr[col + 1] - indptr[col];
    int64_t fanout = sub_indptr[row + 1] - sub_indptr[row];
    int64_t tid = threadIdx.x;
    while (tid < fanout) {
      // Sequential Sampling
      // const int64_t edge = tid % degree;
      // Random Sampling
      const int64_t edge = hiprand(&rng) % degree;
      sub_indices[out_start + tid] = indices[in_start + edge];
      tid += blockDim.x;
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
torch::Tensor SampleSubIndicesFused(torch::Tensor indptr, torch::Tensor indices,
                                    torch::Tensor sub_indptr,
                                    torch::Tensor column_ids, bool replace) {
  // nvtxRangePush(__FUNCTION__);
  // nvtxMark("==SampleSubIndicesFused==");
  int64_t size = sub_indptr.numel() - 1;
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(sub_indptr.data_ptr<IdType>()));
  int n_edges = item_prefix[size];  // cpu
  auto sub_indices = torch::zeros(n_edges, indices.options());

  if (replace) {
    dim3 block(32, 8);
    dim3 grid((size + block.x - 1) / block.x);
    _SampleSubIndicesKernelFusedWithReplace<int64_t><<<grid, block>>>(
        sub_indices.data_ptr<int64_t>(), indptr.data_ptr<int64_t>(),
        indices.data_ptr<int64_t>(), sub_indptr.data_ptr<int64_t>(),
        column_ids.data_ptr<int64_t>(), size);
  } else {
    std::cerr << "Not implemented warning";
  }
  // nvtxRangePop();
  return sub_indices;
}

// Fused columnwise slicing and sampling
std::pair<torch::Tensor, torch::Tensor>
CSCColumnwiseFusedSlicingAndSamplingCUDA(torch::Tensor indptr,
                                         torch::Tensor indices,
                                         torch::Tensor column_ids,
                                         int64_t fanout, bool replace) {
  auto sub_indptr =
      GetSampledSubIndptrFused<int64_t>(indptr, column_ids, fanout, replace);
  auto sub_indices = SampleSubIndicesFused<int64_t>(indptr, indices, sub_indptr,
                                                    column_ids, replace);
  return {sub_indptr, sub_indices};
}

}  // namespace impl
}  // namespace gs