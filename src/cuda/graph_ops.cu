#include "hip/hip_runtime.h"
#include "graph_ops.h"

#include "atomic.h"
#include "cuda_common.h"
#include "utils.h"

namespace gs {
namespace impl {
template <typename IdType, typename DType>
__global__ void GroupSum(IdType* indptr, DType* data, DType* out_data,
                         int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  while (row < size) {
    IdType start = indptr[row];
    IdType n_edges = indptr[row + 1] - start;
    for (int idx = threadIdx.x; idx < n_edges; idx += blockDim.x) {
      AtomicAdd(&out_data[row], data[start + idx]);
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType, typename DType>
__global__ void GroupDiv(IdType* indptr, DType* data, DType* divisor,
                         DType* out_data, int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  while (row < size) {
    IdType start = indptr[row];
    IdType n_edges = indptr[row + 1] - start;
    for (int idx = threadIdx.x; idx < n_edges; idx += blockDim.x) {
      out_data[start + idx] = data[start + idx] / divisor[row];
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType, typename DType>
__global__ void GroupDiv_2index(IdType* indptr, IdType* e_ids, DType* data,
                                DType* divisor, DType* out_data, int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  while (row < size) {
    IdType start = indptr[row];
    IdType n_edges = indptr[row + 1] - start;
    IdType pos;
    for (int idx = threadIdx.x; idx < n_edges; idx += blockDim.x) {
      pos = e_ids[start + idx];
      out_data[pos] = data[pos] / divisor[row];
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType, typename DType>
__global__ void GroupNormL2(IdType* indptr, DType* data, DType* out_data,
                            int64_t size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;

  while (row < size) {
    IdType start = indptr[row];
    IdType n_edges = indptr[row + 1] - start;
    for (int idx = threadIdx.x; idx < n_edges; idx += blockDim.x) {
      AtomicAdd(&out_data[row], powf(data[start + idx], 2));
    }
    row += gridDim.x * blockDim.y;
  }
}

torch::Tensor GraphSumCUDA(torch::Tensor indptr, torch::Tensor data,
                           torch::optional<torch::Tensor> e_ids) {
  auto size = indptr.numel() - 1;
  auto group_sum = torch::zeros(size, data.options());
  dim3 block(32, 16);
  dim3 grid((size + block.x - 1) / block.x);
  GroupSum<int64_t, _Float32>
      <<<grid, block>>>(indptr.data_ptr<int64_t>(), data.data_ptr<_Float32>(),
                        group_sum.data_ptr<_Float32>(), size);
  return group_sum;
}

torch::Tensor GraphL2NormCUDA(torch::Tensor indptr, torch::Tensor data) {
  auto size = indptr.numel() - 1;
  auto group_norm = torch::zeros(size, data.options());
  dim3 block(32, 16);
  dim3 grid((size + block.x - 1) / block.x);
  GroupNormL2<int64_t, _Float32>
      <<<grid, block>>>(indptr.data_ptr<int64_t>(), data.data_ptr<_Float32>(),
                        group_norm.data_ptr<_Float32>(), size);
  return group_norm;
}

torch::Tensor GraphDivCUDA(torch::Tensor indptr, torch::Tensor data,
                           torch::Tensor divisor) {
  auto size = indptr.numel() - 1;
  auto out_data = torch::zeros(data.numel(), data.options());
  dim3 block(32, 16);
  dim3 grid((size + block.x - 1) / block.x);
  GroupDiv<int64_t, _Float32><<<grid, block>>>(
      indptr.data_ptr<int64_t>(), data.data_ptr<_Float32>(),
      divisor.data_ptr<_Float32>(), out_data.data_ptr<_Float32>(), size);
  return out_data;
}

torch::Tensor GraphDivCUDA_2index(torch::Tensor indptr, torch::Tensor data,
                                  torch::Tensor e_ids, torch::Tensor divisor) {
  auto size = indptr.numel() - 1;
  auto out_data = torch::zeros(data.numel(), data.options());
  dim3 block(32, 16);
  dim3 grid((size + block.x - 1) / block.x);
  GroupDiv_2index<int64_t, _Float32>
      <<<grid, block>>>(indptr.data_ptr<int64_t>(), e_ids.data_ptr<int64_t>(),
                        data.data_ptr<_Float32>(), divisor.data_ptr<_Float32>(),
                        out_data.data_ptr<_Float32>(), size);
  return out_data;
}

torch::Tensor GraphNormalizeCUDA(torch::Tensor indptr, torch::Tensor data) {
  torch::Tensor out_data, group_sum;
  group_sum = GraphSumCUDA(indptr, data);
  out_data = GraphDivCUDA(indptr, data, group_sum);
  return out_data;
}
}  // namespace impl
}  // namespace gs