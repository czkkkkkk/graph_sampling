#include "hip/hip_runtime.h"
#include "graph_ops.h"

#include <hiprand/hiprand_kernel.h>
#include "atomic.h"
#include "cuda_common.h"
#include "macro.h"
#include "utils.h"

namespace gs {
namespace impl {
//////////////////////// CSCSumCUDA //////////////////////////
/**
 * @brief SpMV for CSCSum
 */
template <typename IdType, typename DType, bool UseEMap, bool UseNMap>
__global__ void _SegmentSumKernel(IdType* indptr, IdType* EMap, IdType* NMap,
                                  DType* data, int num_rows, int powk,
                                  int out_len, DType* out) {
  // SPMM with CSR.
  int ty = blockIdx.x * blockDim.y + threadIdx.y;
  const IdType stride_y = blockDim.y * gridDim.x;
  const int stride_x = blockDim.x * gridDim.y;
  while (ty < num_rows) {
    int tx = blockIdx.y * blockDim.x + threadIdx.x;
    while (tx < out_len) {
      DType local_accum = 0;
      for (IdType i = indptr[ty]; i < indptr[ty + 1]; ++i) {
        const IdType data_idx = UseEMap ? EMap[i] : i;
        const DType* dataoff = data + data_idx * out_len;
        DType tmp = powk == 1 ? dataoff[tx] : __powf(dataoff[tx], powk);
        local_accum += tmp;
      }
      int out_pos = UseNMap ? NMap[ty * out_len + tx] : ty * out_len + tx;
      out[out_pos] = local_accum;
      tx += stride_x;
    }
    ty += stride_y;
  }
}

/**
 * @brief SpMMCOO for graphSum
 */
template <typename IdType, typename DType, bool UseEMap>
__global__ void _SegmentSumCOOKernel(IdType* target, IdType* EMap, DType* data,
                                     int64_t E, int powk, int out_len,
                                     DType* out) {
  // SPMM with COO.
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  const IdType stride_y = blockDim.y * gridDim.y;
  const int64_t stride_x = blockDim.x * gridDim.x;
  while (ty < E) {
    int64_t tx = blockIdx.x * blockDim.x + threadIdx.x;
    const IdType data_idx = UseEMap ? EMap[ty] : ty;
    const DType* dataoff = data + data_idx * out_len;
    DType* outoff = out + target[ty] * out_len;
    while (tx < out_len) {
      DType val = powk == 1 ? dataoff[tx] : __powf(dataoff[tx], powk);
      AtomicAdd(outoff + tx, val);
      tx += stride_x;
    }
    ty += stride_y;
  }
}

template <typename IdType, typename DType>
void CSCSum(torch::Tensor indptr, torch::optional<torch::Tensor> e_ids,
            torch::optional<torch::Tensor> n_ids, torch::Tensor data,
            torch::Tensor out_data, int64_t powk) {
  auto num_element = indptr.numel() - 1;
  auto use_n_map = n_ids.has_value(), use_e_map = e_ids.has_value();
  auto n_ids_map = use_n_map ? n_ids.value().data_ptr<IdType>() : nullptr;
  auto e_ids_map = use_e_map ? e_ids.value().data_ptr<IdType>() : nullptr;

  // Aligning DGL
  const int out_len = 1;

  const int ntx = 1;
  const int nty = 256;
  const int nby = (out_len + ntx - 1) / ntx;
  const int nbx = (num_element + nty - 1) / nty;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  SWITCH_IDX(use_e_map, use_n_map, {
    CUDA_KERNEL_CALL((_SegmentSumKernel<IdType, DType, UseEMap, UseNMap>),
                     nblks, nthrs, indptr.data_ptr<IdType>(), e_ids_map,
                     n_ids_map, data.data_ptr<DType>(), num_element, powk,
                     out_len, out_data.data_ptr<DType>());
  });
}

void CSCSumCUDA(torch::Tensor indptr, torch::optional<torch::Tensor> e_ids,
                torch::optional<torch::Tensor> n_ids, torch::Tensor data,
                torch::Tensor out_data, int64_t powk) {
  CSCSum<int64_t, float>(indptr, e_ids, n_ids, data, out_data, powk);
}

template <typename IdType, typename DType>
void COOSum(torch::Tensor target, torch::optional<torch::Tensor> e_ids,
            torch::Tensor data, torch::Tensor out_data, int64_t powk) {
  int64_t E = target.numel();
  auto use_e_map = e_ids.has_value();
  auto e_ids_map = use_e_map ? e_ids.value().data_ptr<IdType>() : nullptr;

  // Aligning DGL
  const int out_len = 1;

  const int ntx = FindNumThreads(out_len);
  const int nty = CUDA_MAX_NUM_THREADS / ntx;
  const int nbx = (out_len + ntx - 1) / ntx;
  const int nby = FindNumBlocks<'y'>((E + nty - 1) / nty);
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  SWITCH_IDX(use_e_map, false, {
    CUDA_KERNEL_CALL((_SegmentSumCOOKernel<IdType, DType, UseEMap>), nblks,
                     nthrs, target.data_ptr<IdType>(), e_ids_map,
                     data.data_ptr<DType>(), E, powk, out_len,
                     out_data.data_ptr<DType>());
  });
}

void COOSumCUDA(torch::Tensor target, torch::optional<torch::Tensor> e_ids,
                torch::Tensor data, torch::Tensor out_data, int64_t powk) {
  COOSum<int64_t, float>(target, e_ids, data, out_data, powk);
}

//////////////////////// CSCNormalizeCUDA //////////////////////////
/**
 * @brief SpMV for CSCNormalize
 */
template <typename IdType, typename DType, bool UseEMap>
__global__ void _SegmentNormalizeKernel(IdType* indptr, IdType* EMap,
                                        DType* data, int num_rows, int out_len,
                                        DType* out) {
  // SPMM with CSR.
  int ty = blockIdx.x * blockDim.y + threadIdx.y;
  const IdType stride_y = blockDim.y * gridDim.x;
  const int stride_x = blockDim.x * gridDim.y;
  while (ty < num_rows) {
    int tx = blockIdx.y * blockDim.x + threadIdx.x;
    while (tx < out_len) {
      DType local_accum = 0;
      for (IdType i = indptr[ty]; i < indptr[ty + 1]; ++i) {
        const IdType data_idx = UseEMap ? EMap[i] : i;
        const DType* dataoff = data + data_idx * out_len;
        local_accum += dataoff[tx];
      }
      for (IdType i = indptr[ty]; i < indptr[ty + 1]; ++i) {
        const IdType data_idx = UseEMap ? EMap[i] : i;
        const DType* indataoff = data + data_idx * out_len;
        DType* outdataoff = out + data_idx * out_len;
        outdataoff[tx] = indataoff[tx] / local_accum;
      }
      tx += stride_x;
    }
    ty += stride_y;
  }
}

template <typename IdType, typename DType>
void CSCNormalize(torch::Tensor indptr, torch::optional<torch::Tensor> e_ids,
                  torch::Tensor data, torch::Tensor out_data) {
  auto num_element = indptr.numel() - 1;
  auto use_e_map = e_ids.has_value();
  auto e_ids_map = use_e_map ? e_ids.value().data_ptr<IdType>() : nullptr;

  // Aligning DGL
  const int out_len = 1;

  const int ntx = 1;
  const int nty = 256;
  const int nby = (out_len + ntx - 1) / ntx;
  const int nbx = (num_element + nty - 1) / nty;
  const dim3 nblks(nbx, nby);
  const dim3 nthrs(ntx, nty);
  SWITCH_IDX(use_e_map, false, {
    CUDA_KERNEL_CALL((_SegmentNormalizeKernel<IdType, DType, UseEMap>), nblks,
                     nthrs, indptr.data_ptr<IdType>(), e_ids_map,
                     data.data_ptr<DType>(), num_element, out_len,
                     out_data.data_ptr<DType>());
  });
}

void CSCNormalizeCUDA(torch::Tensor indptr,
                      torch::optional<torch::Tensor> e_ids, torch::Tensor data,
                      torch::Tensor out_data) {
  CSCNormalize<int64_t, float>(indptr, e_ids, data, out_data);
}

template <typename IdType>
__global__ void _CSCSplitKernel(IdType* indptr, IdType* out, IdType* out_sizes,
                                int64_t num_split, int64_t split_size,
                                int64_t last_size) {
  int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
  while (row < num_split) {
    int64_t inoff = row * split_size;
    int64_t outoff = row * (split_size + 1);
    int64_t size = (row == num_split - 1) ? last_size : split_size;
    IdType prefix = indptr[inoff];
    out[outoff] = 0;
    out_sizes[row] = indptr[inoff + size] - prefix;
    for (int idx = threadIdx.x; idx < size; idx += blockDim.x) {
      out[outoff + idx + 1] = indptr[inoff + idx + 1] - prefix;
    }
    row += gridDim.x * blockDim.y;
  }
}

template <typename IdType>
std::vector<std::vector<torch::Tensor>> _CSCSplit(
    torch::Tensor indptr, torch::Tensor indices,
    torch::optional<torch::Tensor> eid, int64_t split_size) {
  std::vector<torch::Tensor> sub_indptrs;
  std::vector<torch::Tensor> sub_indices;
  std::vector<torch::Tensor> sub_eids;

  auto total_element = indptr.numel() - 1;
  auto num_split = total_element / split_size;
  auto redundant = total_element - num_split * split_size;
  int64_t total_indptrs_len;
  if (redundant != 0) {
    total_indptrs_len = num_split * (split_size + 1) + redundant + 1;
    num_split += 1;
  } else {
    total_indptrs_len = num_split * (split_size + 1);
    redundant = split_size;
  }
  auto total_indptrs = torch::empty(total_indptrs_len, indptr.options());
  auto indices_split_sizes = torch::empty(num_split, indices.options());

  dim3 block(64, 8);
  dim3 grid((num_split + block.y - 1) / block.y);
  CUDA_KERNEL_CALL((_CSCSplitKernel<IdType>), grid, block,
                   indptr.data_ptr<IdType>(), total_indptrs.data_ptr<IdType>(),
                   indices_split_sizes.data_ptr<IdType>(), num_split,
                   split_size, redundant);

  sub_indptrs = torch::split(total_indptrs, split_size + 1);

  indices_split_sizes = indices_split_sizes.to(torch::kCPU);
  auto data_ptr = indices_split_sizes.data_ptr<IdType>();
  std::vector<IdType> subindices_sizes(data_ptr, data_ptr + num_split);
  sub_indices = torch::split_with_sizes(indices, subindices_sizes);
  if (eid.has_value())
    sub_eids = torch::split_with_sizes(eid.value(), subindices_sizes);

  return {sub_indptrs, sub_indices, sub_eids};
}

std::vector<std::vector<torch::Tensor>> CSCSplitCUDA(
    torch::Tensor indptr, torch::Tensor indices,
    torch::optional<torch::Tensor> eid, int64_t split_size) {
  return _CSCSplit<int64_t>(indptr, indices, eid, split_size);
}
}  // namespace impl
}  // namespace gs